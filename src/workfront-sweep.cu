#include "hip/hip_runtime.h"
#include "workfront-sweep.cuh"
#include <cub/device/device_select.cuh>
#include <cub/block/block_scan.cuh>

enum OutType {
    QUEUE,
    TOUCHED,
    FRONTIER
};


enum CoopType{
    VANILLA,
    FULL
};

template <OutType out_type>
__global__ void wf_iter_simple(CSRGraph g, edge_data_type* d, index_type* last_q, index_type last_q_len, index_type* out, index_type* pq_idx, index_type* scratch) {
    index_type index = threadIdx.x + (blockDim.x * blockIdx.x);

    if (index < last_q_len) {
        index_type s_idx = last_q[index];
        // somewhat baed on https://towardsdatascience.com/bellman-ford-single-source-shortest-path-algorithm-on-gpu-using-cuda-a358da20144b
        for (int j = g.row_start[s_idx]; j < g.row_start[s_idx + 1]; j++) {
            edge_data_type w = d[s_idx];
            edge_data_type ew = g.edge_data[j];
            index_type n = g.edge_dst[j];
            edge_data_type nw = d[n];
            edge_data_type new_w = ew + w;
            // Check if the distance is already set to max then just take the max since,
            if (w >= MAX_VAL){
                new_w = MAX_VAL;
            }

            //printf("source %u, dst %u, new_w %u\n", s_idx, n, new_w);

            if (new_w < nw) {
                atomicMin(&d[n],new_w);
                if (out_type == OutType::QUEUE) {
                    if (atomicCAS(&scratch[n],0,index) == 0) {
                        index_type q_idx = atomicAdd(pq_idx,1);
                        out[q_idx] = n;
                    }
                } else if (out_type == OutType::TOUCHED) {
                    out[n] = 1;
                }
            }
        }
    }
}

template <int block_size>
double wf_sweep_atomicq(CSRGraph& g, CSRGraph& d_g, edge_data_type* d_dists, index_type source, bool verbose=false) {
    double start,end = 0;
    index_type* q1, *q2 = NULL;
    index_type* qscratch = NULL;
    check_cuda(hipMalloc(&q1, g.nnodes * sizeof(index_type)));
    check_cuda(hipMalloc(&q2, g.nnodes * sizeof(index_type)));
    check_cuda(hipMalloc(&qscratch, g.nnodes* sizeof(index_type)));
    // Set first q entry to 0 (source) TODO: other sources
    check_cuda(hipMemcpy(q1, &source, sizeof(index_type), hipMemcpyHostToDevice));
    index_type* qlen = NULL;
    check_cuda(hipMallocManaged(&qlen, sizeof(index_type)));
    *qlen = 1;

    //index_type* hq = NULL;
    //hipHostAlloc(&hq,g.nnodes*sizeof(index_type),hipHostMallocDefault);

    start = getTimeStamp();

    int itr = 0;
    while (*qlen) {
        if (verbose) {
            printf("Iter %d, qlen %d\n",itr, *qlen);
        }
        index_type len = *qlen;
        *qlen = 0;
 
        wf_iter_simple<OutType::QUEUE><<<(len + block_size - 1) / block_size,block_size>>>(d_g, d_dists, q1, len,q2, qlen, qscratch);
        check_cuda(hipMemset(qscratch,0,g.nnodes*sizeof(index_type)));
        hipDeviceSynchronize();

        index_type* tmp = q1;
        q1 = q2;
        q2 = tmp;


        itr += 1;
    }
    end = getTimeStamp();
    double gpu_time = end - start;
    if (verbose) printf("GPU time: %f\n",gpu_time);


    check_cuda(hipFree(q1));
    check_cuda(hipFree(q2));
    check_cuda(hipFree(qscratch));
    check_cuda(hipFree(qlen));
    return gpu_time;
}

///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

template <int block_size, OutType out_type>
__global__ void wf_coop_iter_impl1(CSRGraph g, edge_data_type* d, index_type* last_q, index_type last_q_len, index_type* out, index_type* pq_idx,  index_type* scratch) {
    index_type index = threadIdx.x + (blockDim.x * blockIdx.x);

    __shared__ index_type block_row_start;
    __shared__ index_type block_row_end; 
    __shared__ index_type s_idx; 

    //decide what each block work range is 
    index_type block_index_start = blockIdx.x * blockDim.x;
    index_type block_index_end = block_index_start + blockDim.x;  
    //do not go beyond last_q_len
    block_index_end = min(block_index_end, last_q_len); 

    for (index_type index=block_index_start; index < block_index_end; index++){
        //only first thread in block load row start and end and source index 
        if (threadIdx.x == 0){
            s_idx = last_q[index];
            block_row_start = g.row_start[s_idx]; 
            block_row_end = g.row_start[s_idx + 1]; 
            //printf("source id %d, block_row_start %d, block_row_end %d \n", s_idx, block_row_start, block_row_end);
        }
        __syncthreads();

        //the threads in this block each take one edge
        for (index_type j =threadIdx.x + block_row_start; j < block_row_end; j += blockDim.x){
            
            edge_data_type w = d[s_idx];
            edge_data_type ew = g.edge_data[j];
            index_type n = g.edge_dst[j];
            edge_data_type nw = d[n];
            edge_data_type new_w = ew + w;
            // Check if the distance is already set to max then just take the max since,
            if (w >= MAX_VAL){
                new_w = MAX_VAL;
            }

            if (new_w < nw) {

                atomicMin(&d[n],new_w);
                if (out_type == OutType::QUEUE){
                    if (atomicCAS(&scratch[n],0,1) == 0) {
                        index_type q_idx = atomicAdd(pq_idx,1);
                        out[q_idx] = n;
                    }
                }
                else if (out_type == OutType::TOUCHED){
                    out[n] = 1;
                }
            }
        }

    }
    
}



__device__ index_type bisect_right(index_type *block, index_type lo, index_type hi, index_type value){

    if (value >= block[hi - 1])
        return hi - 1; // if all elemenst in the scan result are smaller than value, the source vertex is the last one 

    index_type mid;
    while (lo < hi){
        mid = lo + (hi - lo) / 2; 
        if (block[mid] > value)
            hi = mid;
        else
            lo = mid + 1;
    }
    return (lo > 0) ? lo - 1 : lo;
}



template <int block_size, OutType out_type>
__global__ void wf_coop_iter_impl2(CSRGraph g, edge_data_type* d, index_type* last_q, index_type last_q_len, index_type* out, index_type* pq_idx,  index_type* scratch, index_type iter) {
    
    //one for start, one for end.
    __shared__ index_type source_vertices[block_size];
    __shared__ index_type offset_start[block_size];
    __shared__ index_type num_neighbors[block_size]; 

    __shared__ index_type total_neighbors;

    // Specialize BlockScan for a 1D block of block_size threads on type int
    typedef hipcub::BlockScan<index_type, block_size> BlockScan;
     
    // Allocate shared memory for BlockScan
    __shared__ typename BlockScan::TempStorage temp_storage;


    index_type global_index = threadIdx.x + (blockDim.x * blockIdx.x);
    index_type local_index = threadIdx.x; 


    //initialize source vertices, else bisect wont work properly
    source_vertices[local_index] = 0; 
    // num_neighbors[local_index] = 0;
    //offset_start[local_index] = 0;
    

    //decide what each block work range is 
    index_type block_index_start = blockIdx.x * blockDim.x;
    index_type block_index_end = block_index_start + blockDim.x;  
    //do not go beyond last_q_len
    block_index_end = min(block_index_end, last_q_len); 


    index_type num_neighbors_local = 0;
    if (global_index < block_index_end){
        //load start and end offset and number of neighbors into shared memory
        if (out_type != TOUCHED) {
            index_type s_idx = last_q[global_index];
            source_vertices[local_index] = s_idx; 
            index_type row_start = g.row_start[s_idx];
            num_neighbors_local = g.row_start[s_idx + 1] - row_start;
            offset_start[local_index] = row_start;
        } else {
            index_type s_idx = global_index;
            if (last_q[s_idx] == iter -1 ) {
                // printf("Found vertex %d\n",s_idx);
                source_vertices[local_index] = s_idx;
                index_type row_start = g.row_start[s_idx];
                num_neighbors_local = g.row_start[s_idx + 1] - row_start;
                offset_start[local_index] = row_start;
            }
        }

        //printf("block_idx %d, global_index %d, block_end %d, num_neighbors %d, source %d\n", blockIdx.x, global_index, block_index_end, num_neighbors[local_index], s_idx);
        
    }
    __syncthreads();

    /****************************** Replace with scan ********************************************************/
    //add total num_neighbors to determine total work, replace with block level exclusive scan also get sum
        // __syncthreads();

    BlockScan(temp_storage).ExclusiveSum(num_neighbors_local, num_neighbors_local, total_neighbors);
    // output_offset[tidx] = degree;
    // if (global_index == 0) {
    // printf("Block aggregate %d\n",total_neighbors);
    // }
    num_neighbors[local_index] = num_neighbors_local;
    // __syncthreads();
    if (local_index == 0 && out_type == TOUCHED) {
        atomicAdd(pq_idx,total_neighbors);
    }



    // if (local_index == 0){
    //     total_neighbors = num_neighbors[0];
    //     num_neighbors[0] = 0; 
    //     index_type temp = 0; 
    //     for (int i = 1; i < block_size; i++){
    //         temp = num_neighbors[i];
    //         num_neighbors[i] = total_neighbors;
    //         total_neighbors += temp;
    //     }

    //     //printf("block_id %d, total neighbors per block %d \n", blockIdx.x, total_neighbors);
    // }

    __syncthreads(); 
    
    /*********************************************************************************************************/
    //each take on a task
    for (index_type work_index = local_index; work_index < total_neighbors; work_index += block_size){

        //find shared mem index, so we can find source, offset start, and degree
        
        index_type shared_mem_index = bisect_right(num_neighbors, 0, block_size, work_index);
        index_type source = source_vertices[shared_mem_index];
        index_type edge_index = offset_start[shared_mem_index] + work_index - num_neighbors[shared_mem_index];

        
        //rest of code remains the same 
        edge_data_type w = d[source];
        edge_data_type ew = g.edge_data[edge_index];
        index_type n = g.edge_dst[edge_index];
        edge_data_type nw = d[n];
        edge_data_type new_w = ew + w;

        // Check if the distance is already set to max then just take the max since,
        if (w >= MAX_VAL){
            new_w = MAX_VAL;
        }

       //printf("local_index %u, worker_index %u, shared_mem_index %u, source %u, dst %d, new_w %d\n", local_index, work_index, shared_mem_index, source, n, new_w);

        if (new_w < nw) {
            atomicMin(&d[n],new_w);
            if (out_type == OutType::QUEUE){
                if (atomicCAS(&scratch[n],0,1) == 0) {
                    index_type q_idx = atomicAdd(pq_idx,1);
                    out[q_idx] = n;
                }
            }
            else if (out_type == OutType::TOUCHED){
                out[n] = iter;
            }
        }

         
    }
    
}


template <int block_size, CoopType coop_impl>
double wf_sweep_coop(CSRGraph& g, CSRGraph& d_g, edge_data_type* d_d, index_type source, bool verbose=false) {
    double start,end = 0;
    index_type* q1, *q2 = NULL;
    index_type* qscratch = NULL;
    check_cuda(hipMalloc(&q1, g.nnodes * sizeof(index_type)));
    check_cuda(hipMalloc(&q2, g.nnodes * sizeof(index_type)));
    check_cuda(hipMalloc(&qscratch, g.nnodes* sizeof(index_type)));
    // Set first q entry to source
    check_cuda(hipMemcpy(q1, &source, sizeof(index_type), hipMemcpyHostToDevice));
    index_type* qlen = NULL;
    check_cuda(hipMallocManaged(&qlen, sizeof(index_type)));
    *qlen = 1;

    //index_type* hq = NULL;
    //hipHostAlloc(&hq,g.nnodes*sizeof(index_type),hipHostMallocDefault);

    start = getTimeStamp();

    int itr = 0;
    while (*qlen) {
        if (verbose) {
            printf("Iter %d, qlen %d\n",itr, *qlen);
        }
        index_type len = *qlen;
        *qlen = 0;

        if (coop_impl == CoopType::VANILLA)
            wf_coop_iter_impl1<block_size, OutType::QUEUE><<<(len + block_size - 1) / block_size,block_size>>>(d_g, d_d, q1, len,q2, qlen, qscratch);
        else if (coop_impl == CoopType::FULL)
            wf_coop_iter_impl2<block_size, OutType::QUEUE><<<(len + block_size - 1) / block_size, block_size>>>(d_g, d_d, q1, len,q2, qlen, qscratch,0);
        check_cuda(hipMemset(qscratch,0,g.nnodes*sizeof(index_type)));
        hipDeviceSynchronize();

        index_type* tmp = q1;
        q1 = q2;
        q2 = tmp;


        itr += 1;
    }
    end = getTimeStamp();
    double gpu_time = end - start;
    if (verbose) printf("GPU time: %f\n",gpu_time);

    check_cuda(hipFree(q1));
    check_cuda(hipFree(q2));
    check_cuda(hipFree(qscratch));
    check_cuda(hipFree(qlen));

    return gpu_time;
}




__global__ void setup_id(index_type* out, index_type n) {
    index_type index = threadIdx.x + (blockDim.x * blockIdx.x);
    if (index < n)  out[index] = index;
}



template <int block_size, CoopType coop_impl>
double wf_coop_filter(CSRGraph& g, CSRGraph& d_g, edge_data_type* d_dists, index_type source, bool verbose=false) {
    double start,end = 0;
    index_type* q = NULL;
    index_type* scan_indices = NULL;
    index_type* touched = NULL;
    check_cuda(hipMalloc(&q, g.nnodes * sizeof(index_type)));
    check_cuda(hipMalloc(&touched, g.nnodes* sizeof(index_type)));
    check_cuda(hipMalloc(&scan_indices, g.nnodes* sizeof(index_type)));
    setup_id<<<(g.nnodes + block_size - 1),block_size>>>(scan_indices,g.nnodes);

    // Set first q entry to source
    check_cuda(hipMemcpy(q, &source, sizeof(index_type),hipMemcpyHostToDevice));
    index_type* qlen = NULL;
    check_cuda(hipMallocManaged(&qlen, sizeof(index_type)));
    *qlen = 1;

    void* flg_tmp_store = NULL;
    size_t flg_store_size = 0;
    // index_type num_selected = 0;
    hipcub::DeviceSelect::Flagged(flg_tmp_store,flg_store_size,scan_indices,touched,q,qlen,g.nnodes);
    check_cuda(hipMalloc(&flg_tmp_store,flg_store_size));


    hipMemset(&q[1],0xFF,(g.nnodes-1)*sizeof(index_type));
    hipMemset(&touched[0],0xFF,(g.nnodes)*sizeof(index_type));

    start = getTimeStamp();
    int iter = 0;
    while (*qlen) {
        if (verbose) {
            printf("Iter %d\n",*qlen);
        }
        index_type len = *qlen;
        *qlen = 0;

        if (coop_impl == CoopType::VANILLA)
            wf_coop_iter_impl1<block_size, OutType::TOUCHED><<<(len + block_size - 1) / block_size, block_size>>>(d_g, d_dists, q, len, touched,qlen, NULL);
        else if (coop_impl == CoopType::FULL)
            wf_coop_iter_impl2<block_size, OutType::TOUCHED><<<(g.nnodes + block_size - 1) / block_size, block_size>>>(d_g, d_dists, q, g.nnodes, touched,qlen, NULL,iter+1);
        // hipcub::DeviceSelect::Flagged(flg_tmp_store,flg_store_size,scan_indices,touched,q,qlen,g.nnodes);
        // check_cuda(hipMemset(touched,0,g.nnodes*sizeof(index_type)));
        hipDeviceSynchronize();
        index_type* tmp = q;
        q = touched;
        touched = tmp;
        iter++;

        // printf("res: %d\n",*qlen);
    }
    end = getTimeStamp();
    double gpu_time = end - start;
    if (verbose) printf("GPU time: %f\n",gpu_time);

    check_cuda(hipFree(q));
    check_cuda(hipFree(touched));
    check_cuda(hipFree(scan_indices));
    check_cuda(hipFree(qlen));
    return gpu_time;
}


/////////////////////////////////////////////////////////////////////////////////////////////////////////////////
template <int block_size>
double wf_sweep_filter(CSRGraph& g, CSRGraph& d_g, edge_data_type* d_dists, index_type source, bool verbose=false) {
    double start,end = 0;
    index_type* q = NULL;
    index_type* scan_indices = NULL;
    index_type* touched = NULL;
    check_cuda(hipMalloc(&q, g.nnodes * sizeof(index_type)));
    check_cuda(hipMalloc(&touched, g.nnodes* sizeof(index_type)));
    check_cuda(hipMalloc(&scan_indices, g.nnodes* sizeof(index_type)));
    setup_id<<<(g.nnodes + block_size - 1),block_size>>>(scan_indices,g.nnodes);

    // Set first q entry to source
    check_cuda(hipMemcpy(q, &source, sizeof(index_type),hipMemcpyHostToDevice));
    index_type* qlen = NULL;
    check_cuda(hipMallocManaged(&qlen, sizeof(index_type)));
    *qlen = 1;

    void* flg_tmp_store = NULL;
    size_t flg_store_size = 0;
    // index_type num_selected = 0;
    hipcub::DeviceSelect::Flagged(flg_tmp_store,flg_store_size,scan_indices,touched,q,qlen,g.nnodes);
    check_cuda(hipMalloc(&flg_tmp_store,flg_store_size));


    start = getTimeStamp();
    while (*qlen) {
        if (verbose) {
            printf("Iter %d\n",*qlen);
        }
        index_type len = *qlen;
        *qlen = 0;
 
        wf_iter_simple<OutType::TOUCHED><<<(len + block_size - 1) / block_size,block_size>>>(d_g, d_dists, q, len, touched,NULL, NULL);
        //coop example, interface exactly the same 
        //wf_coop_iter_impl2<block_size, OutType::TOUCHED><<<(len + block_size - 1) / block_size, block_size>>>(d_g, d_dists, q, len, touched,NULL, NULL);
        hipcub::DeviceSelect::Flagged(flg_tmp_store,flg_store_size,scan_indices,touched,q,qlen,g.nnodes);
        check_cuda(hipMemset(touched,0,g.nnodes*sizeof(index_type)));
        hipDeviceSynchronize();
    }
    end = getTimeStamp();
    double gpu_time = end - start;
    if (verbose) printf("GPU time: %f\n",gpu_time);

    check_cuda(hipFree(q));
    check_cuda(hipFree(touched));
    check_cuda(hipFree(scan_indices));
    check_cuda(hipFree(qlen));
    return gpu_time;
}

///////////////////////////////////////////////////////////////////////////////////////////

template <int block_size, OutType out_type>
__global__ void wf_frontier_kernel(CSRGraph g, edge_data_type* d, index_type* frontier_in, index_type* out, index_type n, index_type* out_size, index_type* scratch, index_type iter) {
    __shared__ index_type vertices[block_size];
    __shared__ index_type first_edge_offset[block_size];
    __shared__ index_type output_offset[block_size];
    __shared__ uint64_t block_offset[1];

    // Specialize BlockScan for a 1D block of 128 threads on type int
    typedef hipcub::BlockScan<index_type, block_size> BlockScan;
     
    // Allocate shared memory for BlockScan
    __shared__ typename BlockScan::TempStorage temp_storage;

    index_type gidx = threadIdx.x + (blockDim.x * blockIdx.x);
    index_type tidx = threadIdx.x;
    
    index_type degree = 0;
    if (gidx < n) {
        if (out_type == OutType::FRONTIER) {
            index_type v = frontier_in[gidx];
            if (v != UINT_MAX) {
                index_type row_start =  g.row_start[v];
                degree = g.row_start[v+1] - row_start;
                first_edge_offset[tidx] = row_start;
            }
            vertices[tidx] = v;
        } else {
            index_type v = gidx;
            if (frontier_in[v] == iter - 1) {
                index_type row_start =  g.row_start[v];
                degree = g.row_start[v+1] - row_start;
                first_edge_offset[tidx] = row_start;
            } else {
                v = UINT_MAX;
            }
            vertices[tidx] = v;

        }
    }

    // if (gidx == 0) {
    // printf("Thread %d has node %d with degree %d\n",tidx, vertices[tidx], degree);
    // }

    __syncthreads();

    index_type block_aggregate = 0;
    BlockScan(temp_storage).ExclusiveSum(degree, degree, block_aggregate);
    output_offset[tidx] = degree;
    // if (gidx == 0) {
    // printf("Block aggregate %d\n",block_aggregate);
    // }
    __syncthreads();

    // if (out_type == OutType::FRONTIER) {
        if (tidx == 0 && block_aggregate) {
            // printf("adding %d to %d\n",block_aggregate, *out_size);
            block_offset[0] = atomicAdd(out_size,block_aggregate);
        }
    // }
    // if (gidx == 0) {
    // printf("\nBlock totals %d\n",*out_size);
    // }


    __syncthreads();

    for (index_type edge_id = tidx; edge_id < block_aggregate; edge_id += block_size) {
        // search for edge
        index_type v_id = 0;
        {
            index_type lo = 0;
            index_type hi = block_size;
            while (lo != hi-1) {
                v_id = lo + (hi - lo)/2;
                if (edge_id >= output_offset[v_id]) {
                    lo = v_id;
                } else {
                    hi = v_id;
                }
            }
            v_id = lo;
        }

        index_type edge_offset = edge_id - output_offset[v_id];
        index_type v_in = vertices[v_id];
        // if (v_in == 0) printf("Got vertex 0! Edge_id %d\n", edge_id);
        index_type edge_dst = g.edge_dst[first_edge_offset[v_id]+ edge_offset];
        edge_data_type ew = g.edge_data[first_edge_offset[v_id]+ edge_offset];
        // printf("exploring edge %d\n",first_edge_offset[v_id]+ edge_offset);
        edge_data_type vw = d[v_in];
        edge_data_type old_dw = d[edge_dst];
        edge_data_type new_dw = vw + ew;

        if (out_type == OutType::FRONTIER) {
            index_type out_val = UINT_MAX;
            if (new_dw < old_dw) {
                atomicMin(&d[edge_dst], new_dw);
                out_val = edge_dst;
            }
            out[block_offset[0] + edge_id] = out_val;
        } else if (out_type == OutType::TOUCHED) {
            if (new_dw < old_dw) {
                atomicMin(&d[edge_dst], new_dw);           
                out[edge_dst] = iter;
            }
        } else if (out_type == OutType::QUEUE) {
            if (new_dw < old_dw) {
                atomicMin(&d[edge_dst], new_dw);
                if (atomicCAS(&scratch[edge_dst],0,gidx) == 0) {
                    index_type q_idx = atomicAdd(out_size,1);
                    out[q_idx] = edge_dst;
                }
            }
        }




                //         atomicMin(&d[n],new_w);
                // if (out_type == OutType::QUEUE) {
                //     if (atomicCAS(&scratch[n],0,index) == 0) {
                //         index_type q_idx = atomicAdd(pq_idx,1);
                //         out[q_idx] = n;
                //     }
                // } else if (out_type == OutType::TOUCHED) {
                //     out[n] = 1;
                // }
        // } else {
        //     frontier_out[block_offset[0] + edge_id] = UINT_MAX;
        // }
        // if (d[edge_dst] == 0) printf("\nEdge ID %d. Using local vert %d (%d)source weight: %d. Dest: %d (%d). new: %d \n", edge_id, v_id, v_in, vw, edge_dst, old_dw, new_dw);

    }
}

__global__ void filter_frontier(index_type* frontier_in, index_type* frontier_out, index_type n, index_type* visited, index_type iteration) {
    index_type gidx = threadIdx.x + (blockDim.x * blockIdx.x);

    if (gidx < n) {
        index_type v = frontier_in[gidx];
        index_type out = UINT_MAX;
        if (v != UINT_MAX) {
            if (atomicExch(&visited[v],iteration) != iteration) {
                out = v;
            }
        }
        frontier_out[gidx] = out;
    }
}

template <int block_size, OutType out_type>
double wf_sweep_frontier(CSRGraph& g, CSRGraph& d_g, edge_data_type* d_dists, index_type source, bool verbose=false) {
    double start,end = 0;
    index_type* frontier1, *frontier2 = NULL;
    check_cuda(hipMalloc(&frontier1, g.nedges * sizeof(index_type)));
    check_cuda(hipMalloc(&frontier2, g.nedges * sizeof(index_type)));
    index_type* m_N = NULL;
    index_type* visited = NULL;
    check_cuda(hipMalloc(&visited, g.nnodes * sizeof(index_type)));
    check_cuda(hipMallocManaged(&m_N, sizeof(index_type)));
    *m_N = 1;
    check_cuda(hipMemcpy(frontier1,&source, sizeof(index_type), hipMemcpyHostToDevice));

    index_type* scan_indices = NULL;
    void* flg_tmp_store = NULL;
    size_t flg_store_size = 0;
    if (out_type == OutType::TOUCHED) {
        check_cuda(hipMalloc(&scan_indices, g.nnodes* sizeof(index_type)));
        setup_id<<<(g.nnodes + block_size - 1),block_size>>>(scan_indices,g.nnodes);
        // index_type num_selected = 0;
        hipcub::DeviceSelect::Flagged(flg_tmp_store,flg_store_size,scan_indices,frontier2,frontier1,m_N,g.nnodes);
        check_cuda(hipMalloc(&flg_tmp_store,flg_store_size));


        hipMemset(&frontier1[1], 0xFF, (g.nnodes-1)*sizeof(index_type));
        hipMemset(&frontier2[0], 0xFF, (g.nnodes)*sizeof(index_type));
    }

    start = getTimeStamp();

    index_type iter = 0;
    while(*m_N) {
        index_type n = *m_N;
        *m_N = 0;
        if (verbose) {
            printf("Iter %d\n",n);
        }

        int grid = (((out_type == OutType::TOUCHED)?g.nnodes:n) + block_size-1)/block_size;
        wf_frontier_kernel<block_size, out_type><<<grid,block_size>>>(d_g, d_dists, frontier1, frontier2, (out_type != OutType::TOUCHED)?n:g.nnodes, m_N, NULL, iter+1);
        hipDeviceSynchronize();


        // filter
        if (out_type == OutType::FRONTIER) {
            n = *m_N;
            filter_frontier<<<(n + block_size-1)/block_size,block_size>>>(frontier2, frontier1,n,visited,iter+1);
            hipDeviceSynchronize();
        } else if (out_type == OutType::TOUCHED) {
            // hipcub::DeviceSelect::Flagged(flg_tmp_store,flg_store_size,scan_indices,frontier2,frontier1,m_N,g.nnodes);
            // check_cuda(hipMemset(frontier2,0,g.nnodes*sizeof(index_type)));
            // hipDeviceSynchronize();

            index_type* tmp = frontier1;
            frontier1 = frontier2;
            frontier2 = tmp;
        }


        iter++;
    }
    end = getTimeStamp();
    double gpu_time = end - start;
    if (verbose) printf("GPU time: %f\n",gpu_time);

    check_cuda(hipFree(frontier1));
    check_cuda(hipFree(frontier2));
    check_cuda(hipFree(visited));
    check_cuda(hipFree(m_N));

    return gpu_time;
}



/////////////////////////////////////////////////////////

void initialize_dists(edge_data_type* d_d, index_type n, index_type source) {
    check_cuda(hipMemset(&d_d[0], 0xFF,  n * sizeof(edge_data_type)));
    check_cuda(hipMemset(&d_d[source], 0,  sizeof(edge_data_type)));
}

void workfront_sweep(CSRGraph& g, edge_data_type* dists, index_type source) {
    CSRGraph d_g;
    g.copy_to_gpu(d_g);
    edge_data_type* d_d = NULL;
    check_cuda(hipMalloc(&d_d, g.nnodes * sizeof(edge_data_type)));

    // Initialize for source node
    initialize_dists(d_d, g.nnodes, source);

    // wf_sweep_frontier<256,OutType::TOUCHED>(g, d_g, d_d, source,true);
    wf_coop_filter<32,CoopType::FULL>(g, d_g, d_d, source,true);
    // wf_sweep_frontier<32,OutType::TOUCHED>, "frontier_filt_32" },

    hipMemcpy(dists, d_d, g.nnodes * sizeof(edge_data_type), hipMemcpyDeviceToHost);
}


struct Test {
    double (*f)(CSRGraph&, CSRGraph&, edge_data_type*, index_type, bool);
    const char* name;
};

void workfront_sweep_evaluation(CSRGraph& g, edge_data_type* dists, index_type source, edge_data_type* cpu) {
    CSRGraph d_g;
    g.copy_to_gpu(d_g);
    edge_data_type* d_d = NULL;
    check_cuda(hipMalloc(&d_d, g.nnodes * sizeof(edge_data_type)));
    // Initialize for source node

    Test tests[] = {
        { wf_sweep_atomicq<32>, "atomic_32" },
        { wf_sweep_atomicq<64>, "atomic_64" },
        { wf_sweep_atomicq<128>, "atomic_128" },
        { wf_sweep_atomicq<256>, "atomic_256" },
        { wf_sweep_atomicq<512>, "atomic_512" },
        { wf_sweep_filter<32>, "filter_32" },
        { wf_sweep_filter<64>, "filter_64" },
        { wf_sweep_filter<128>, "filter_128" },
        { wf_sweep_filter<256>, "filter_256" },
        { wf_sweep_filter<512>, "filter_512" },
        { wf_sweep_coop<32,CoopType::VANILLA>, "vanilla_coop_32" },
        { wf_sweep_coop<64,CoopType::VANILLA>, "vanilla_coop_64" },
        { wf_sweep_coop<128,CoopType::VANILLA>, "vanilla_coop_128" },
        { wf_sweep_coop<256,CoopType::VANILLA>, "vanilla_coop_256" },
        { wf_sweep_coop<512,CoopType::VANILLA>, "vanilla_coop_512" },
        { wf_sweep_coop<32,CoopType::FULL>, "coop_32" },
        { wf_sweep_coop<64,CoopType::FULL>, "coop_64" },
        { wf_sweep_coop<128,CoopType::FULL>, "coop_128" },
        { wf_sweep_coop<256,CoopType::FULL>, "coop_256" },
        { wf_sweep_coop<512,CoopType::FULL>, "coop_512" },
        { wf_coop_filter<32,CoopType::FULL>, "coop_filter_32" },
        { wf_coop_filter<64,CoopType::FULL>, "coop_filter_64" },
        { wf_coop_filter<128,CoopType::FULL>, "coop_filter_128" },
        { wf_coop_filter<256,CoopType::FULL>, "coop_filter_256" },
        { wf_coop_filter<512,CoopType::FULL>, "coop_filter_512" },
        { wf_sweep_frontier<32,OutType::FRONTIER>, "frontier_32" },
        { wf_sweep_frontier<64,OutType::FRONTIER>, "frontier_64" },
        { wf_sweep_frontier<128,OutType::FRONTIER>, "frontier_128" },
        { wf_sweep_frontier<256,OutType::FRONTIER>, "frontier_256" },
        { wf_sweep_frontier<512,OutType::FRONTIER>, "frontier_512" },
        { wf_sweep_frontier<32,OutType::TOUCHED>, "frontier_filt_32" },
        { wf_sweep_frontier<64,OutType::TOUCHED>, "frontier_filt_64" },
        { wf_sweep_frontier<128,OutType::TOUCHED>, "frontier_filt_128" },
        { wf_sweep_frontier<256,OutType::TOUCHED>, "frontier_filt_256" },
        { wf_sweep_frontier<512,OutType::TOUCHED>, "frontier_filt_512" },


    };

    printf("\n");
    for (int i = 0; i < sizeof(tests)/sizeof(Test); i++) {
        double best_time = 1000.0;
        printf("%s: ",tests[i].name);
        for (int j = 0; j < 5; j++) {
            initialize_dists(d_d, g.nnodes, source);
            double time = tests[i].f(g, d_g, d_d, source,false);
            printf(" %f, ", time);
            if (time < best_time) best_time = time;
        }
        printf("Best: %f", best_time);
        hipMemcpy(dists, d_d, g.nnodes * sizeof(edge_data_type), hipMemcpyDeviceToHost);
        compare(cpu,dists,g.nnodes);
    }
    printf("\n");


    hipMemcpy(dists, d_d, g.nnodes * sizeof(edge_data_type), hipMemcpyDeviceToHost);
}

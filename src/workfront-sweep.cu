#include "hip/hip_runtime.h"
#include "workfront-sweep.cuh"


__global__ void wf_iter(CSRGraph g, edge_data_type* d, index_type* last_q, index_type last_q_len, index_type* new_q, index_type* pq_idx, index_type* scratch) {
    index_type index = threadIdx.x + (blockDim.x * blockIdx.x);

    if (index < last_q_len) {
        index_type s_idx = last_q[index];
        // somewhat baed on https://towardsdatascience.com/bellman-ford-single-source-shortest-path-algorithm-on-gpu-using-cuda-a358da20144b
        for (int j = g.row_start[s_idx]; j < g.row_start[s_idx + 1]; j++) {
            edge_data_type w = d[s_idx];
            edge_data_type ew = g.edge_data[j];
            index_type n = g.edge_dst[j];
            edge_data_type nw = d[n];
            edge_data_type new_w = ew + w;
            // Check if the distance is already set to max then just take the max since,
            if (w >= MAX_VAL){
                new_w = MAX_VAL;
            }

            if (new_w < nw) {
                atomicMin(&d[n],new_w);
                if (atomicCAS(&scratch[n],0,index) == 0) {
                    index_type q_idx = atomicAdd(pq_idx,1);
                    new_q[q_idx] = n;
                }
            }
        }
    }
}

void workfront_sweep(CSRGraph& g, edge_data_type* dists) {
    double start,end = 0;
    CSRGraph d_g;
    g.copy_to_gpu(d_g);
    edge_data_type* d_d = NULL;
    check_cuda(hipMalloc(&d_d, g.nnodes * sizeof(edge_data_type)));
    // Initialize for source node = 0. Otherwise need to change this
    check_cuda(hipMemset(&d_d[1], 0xFF,  (g.nnodes-1) * sizeof(edge_data_type)));

    index_type* q1, *q2 = NULL;
    index_type* qscratch = NULL;
    check_cuda(hipMalloc(&q1, g.nnodes * sizeof(index_type)));
    check_cuda(hipMalloc(&q2, g.nnodes * sizeof(index_type)));
    check_cuda(hipMalloc(&qscratch, g.nnodes* sizeof(index_type)));
    // Set first q entry to 0 (source) TODO: other sources
    check_cuda(hipMemset(q1, 0, sizeof(index_type)));
    index_type* qlen = NULL;
    check_cuda(hipMallocManaged(&qlen, sizeof(index_type)));
    *qlen = 1;

    //index_type* hq = NULL;
    //hipHostAlloc(&hq,g.nnodes*sizeof(index_type),hipHostMallocDefault);

    start = getTimeStamp();
    while (*qlen) {
        printf("Iter %d\n",*qlen);
        index_type len = *qlen;
        *qlen = 0;
 
        wf_iter<<<(len + 512 - 1) / 512,512>>>(d_g, d_d, q1, len,q2, qlen, qscratch);
        check_cuda(hipMemset(qscratch,0,g.nnodes*sizeof(index_type)));
        hipDeviceSynchronize();

        index_type* tmp = q1;
        q1 = q2;
        q2 = tmp;
    }
    end = getTimeStamp();
    double gpu_time = end - start;
    printf("GPU time: %f\n",gpu_time);

    hipMemcpy(dists, d_d, g.nnodes * sizeof(edge_data_type), hipMemcpyDeviceToHost);
}

#include "hip/hip_runtime.h"
#include "workfront-sweep.cuh"
#include <cub/device/device_select.cuh>
#include <cub/block/block_scan.cuh>


__global__ void wf_iter_aq(CSRGraph g, edge_data_type* d, index_type* last_q, index_type last_q_len, index_type* new_q, index_type* pq_idx, index_type* scratch) {
    index_type index = threadIdx.x + (blockDim.x * blockIdx.x);

    if (index < last_q_len) {
        index_type s_idx = last_q[index];
        // somewhat baed on https://towardsdatascience.com/bellman-ford-single-source-shortest-path-algorithm-on-gpu-using-cuda-a358da20144b
        for (int j = g.row_start[s_idx]; j < g.row_start[s_idx + 1]; j++) {
            edge_data_type w = d[s_idx];
            edge_data_type ew = g.edge_data[j];
            index_type n = g.edge_dst[j];
            edge_data_type nw = d[n];
            edge_data_type new_w = ew + w;
            // Check if the distance is already set to max then just take the max since,
            if (w >= MAX_VAL){
                new_w = MAX_VAL;
            }

            //printf("source %u, dst %u, new_w %u\n", s_idx, n, new_w);

            if (new_w < nw) {
                atomicMin(&d[n],new_w);
                if (atomicCAS(&scratch[n],0,index) == 0) {
                    index_type q_idx = atomicAdd(pq_idx,1);
                    new_q[q_idx] = n;
                }
            }
        }
    }
}

template <int block_size>
void wf_sweep_atomicq(CSRGraph& g, CSRGraph& d_g, edge_data_type* d_dists, index_type source) {
    double start,end = 0;
    index_type* q1, *q2 = NULL;
    index_type* qscratch = NULL;
    check_cuda(hipMalloc(&q1, g.nnodes * sizeof(index_type)));
    check_cuda(hipMalloc(&q2, g.nnodes * sizeof(index_type)));
    check_cuda(hipMalloc(&qscratch, g.nnodes* sizeof(index_type)));
    // Set first q entry to 0 (source) TODO: other sources
    check_cuda(hipMemcpy(q1, &source, sizeof(index_type), hipMemcpyHostToDevice));
    index_type* qlen = NULL;
    check_cuda(hipMallocManaged(&qlen, sizeof(index_type)));
    *qlen = 1;

    //index_type* hq = NULL;
    //hipHostAlloc(&hq,g.nnodes*sizeof(index_type),hipHostMallocDefault);

    start = getTimeStamp();

    int itr = 0;
    while (*qlen) {
        printf("Iter %d, qlen %d\n",itr, *qlen);
        index_type len = *qlen;
        *qlen = 0;
 
        wf_iter_aq<<<(len + block_size - 1) / block_size,block_size>>>(d_g, d_dists, q1, len,q2, qlen, qscratch);
        check_cuda(hipMemset(qscratch,0,g.nnodes*sizeof(index_type)));
        hipDeviceSynchronize();

        index_type* tmp = q1;
        q1 = q2;
        q2 = tmp;


        itr += 1;
    }
    end = getTimeStamp();
    double gpu_time = end - start;
    printf("GPU time: %f\n",gpu_time);
}

#define THREADS_PER_BLOCK 256

__global__ void wf_coop_iter_impl1(CSRGraph g, edge_data_type* d, index_type* last_q, index_type last_q_len, index_type* new_q, index_type* pq_idx,  index_type* scratch) {
    index_type index = threadIdx.x + (blockDim.x * blockIdx.x);

    __shared__ index_type block_row_start;
    __shared__ index_type block_row_end; 
    __shared__ index_type s_idx; 

    //decide what each block work range is 
    index_type block_index_start = blockIdx.x * blockDim.x;
    index_type block_index_end = block_index_start + blockDim.x;  
    //do not go beyond last_q_len
    block_index_end = min(block_index_end, last_q_len); 

    for (index_type index=block_index_start; index < block_index_end; index++){
        //only first thread in block load row start and end and source index 
        if (threadIdx.x == 0){
            s_idx = last_q[index];
            block_row_start = g.row_start[s_idx]; 
            block_row_end = g.row_start[s_idx + 1]; 
            //printf("source id %d, block_row_start %d, block_row_end %d \n", s_idx, block_row_start, block_row_end);
        }
        __syncthreads();

        //the threads in this block each take one edge
        for (index_type j =threadIdx.x + block_row_start; j < block_row_end; j += blockDim.x){
            
            edge_data_type w = d[s_idx];
            edge_data_type ew = g.edge_data[j];
            index_type n = g.edge_dst[j];
            edge_data_type nw = d[n];
            edge_data_type new_w = ew + w;
            // Check if the distance is already set to max then just take the max since,
            if (w >= MAX_VAL){
                new_w = MAX_VAL;
            }

            if (new_w < nw) {
                atomicMin(&d[n],new_w);
                if (atomicCAS(&scratch[n],0,index) == 0) {
                    index_type q_idx = atomicAdd(pq_idx,1);
                    new_q[q_idx] = n;
                }
            }
        }

    }
    
}



__device__ index_type bisect_right(index_type *block, index_type lo, index_type hi, index_type value){

    if (value >= block[hi - 1])
        return hi - 1; // if all elemenst in the scan result are smaller than value, the source vertex is the last one 

    index_type mid;
    while (lo < hi){
        mid = lo + (hi - lo) / 2; 
        if (block[mid] > value)
            hi = mid;
        else
            lo = mid + 1;
    }
    return (lo > 0) ? lo - 1 : lo;
    
    // for (int i = 0; i < hi - 1; i++){
    //     if (block[i] <= value && block[i + 1] > value)
    //         return i; 
    // }
    // return hi - 1; // if all elemenst in the scan result are smaller than value, the source vertex is the last one 
}



template <int block_size>
__global__ void wf_coop_iter_impl2(CSRGraph g, edge_data_type* d, index_type* last_q, index_type last_q_len, index_type* new_q, index_type* pq_idx,  index_type* scratch) {
    
    //one for start, one for end.
    __shared__ index_type source_vertices[block_size];
    __shared__ index_type offset_start[block_size];
    __shared__ index_type num_neighbors[block_size]; 

    __shared__ index_type total_neighbors;



    index_type global_index = threadIdx.x + (blockDim.x * blockIdx.x);
    index_type local_index = threadIdx.x; 


    //initialize source vertices, else bisect wont work properly
    source_vertices[local_index] = 0; 
    num_neighbors[local_index] = 0;
    //offset_start[local_index] = 0;
    

    //decide what each block work range is 
    index_type block_index_start = blockIdx.x * blockDim.x;
    index_type block_index_end = block_index_start + blockDim.x;  
    //do not go beyond last_q_len
    block_index_end = min(block_index_end, last_q_len); 

    if (global_index < block_index_end){
        //load start and end offset and number of neighbors into shared memory
        index_type s_idx = last_q[global_index];
        source_vertices[local_index] = s_idx; 
        offset_start[local_index] = g.row_start[s_idx];
        num_neighbors[local_index] = g.row_start[s_idx + 1] - g.row_start[s_idx]; 

        //printf("block_idx %d, global_index %d, block_end %d, num_neighbors %d, source %d\n", blockIdx.x, global_index, block_index_end, num_neighbors[local_index], s_idx);
        
    }
    __syncthreads();

    /****************************** Replace with scan ********************************************************/
    //add total num_neighbors to determine total work, replace with block level exclusive scan also get sum
    if (local_index == 0){
        total_neighbors = num_neighbors[0];
        num_neighbors[0] = 0; 
        index_type temp = 0; 
        for (int i = 1; i < block_size; i++){
            temp = num_neighbors[i];
            num_neighbors[i] = total_neighbors;
            total_neighbors += temp;
        }

        //printf("block_id %d, total neighbors per block %d \n", blockIdx.x, total_neighbors);
    }

    __syncthreads(); 
    
    /*********************************************************************************************************/
    //each take on a task
    for (index_type work_index = local_index; work_index < total_neighbors; work_index += block_size){

        //find shared mem index, so we can find source, offset start, and degree
        
        index_type shared_mem_index = bisect_right(num_neighbors, 0, block_size, work_index);
        index_type source = source_vertices[shared_mem_index];
        index_type edge_index = offset_start[shared_mem_index] + work_index - num_neighbors[shared_mem_index];

        
        //rest of code remains the same 
        edge_data_type w = d[source];
        edge_data_type ew = g.edge_data[edge_index];
        index_type n = g.edge_dst[edge_index];
        edge_data_type nw = d[n];
        edge_data_type new_w = ew + w;

        // Check if the distance is already set to max then just take the max since,
        if (w >= MAX_VAL){
            new_w = MAX_VAL;
        }

       //printf("local_index %u, worker_index %u, shared_mem_index %u, source %u, dst %d, new_w %d\n", local_index, work_index, shared_mem_index, source, n, new_w);

        if (new_w < nw) {
            atomicMin(&d[n],new_w);
            if (atomicCAS(&scratch[n],0,1) == 0) {
                index_type q_idx = atomicAdd(pq_idx,1);
                new_q[q_idx] = n;
            }
        }

         
    }
    
}




template <int block_size>
void wf_sweep_coop(CSRGraph& g, CSRGraph& d_g, edge_data_type* d_d, index_type source) {
    double start,end = 0;
    index_type* q1, *q2 = NULL;
    index_type* qscratch = NULL;
    check_cuda(hipMalloc(&q1, g.nnodes * sizeof(index_type)));
    check_cuda(hipMalloc(&q2, g.nnodes * sizeof(index_type)));
    check_cuda(hipMalloc(&qscratch, g.nnodes* sizeof(index_type)));
    // Set first q entry to source
    check_cuda(hipMemcpy(q1, &source, sizeof(index_type), hipMemcpyHostToDevice));
    index_type* qlen = NULL;
    check_cuda(hipMallocManaged(&qlen, sizeof(index_type)));
    *qlen = 1;

    //index_type* hq = NULL;
    //hipHostAlloc(&hq,g.nnodes*sizeof(index_type),hipHostMallocDefault);

    start = getTimeStamp();

    int itr = 0;
    while (*qlen) {
        printf("Iter %d, qlen %d\n",itr, *qlen);
        index_type len = *qlen;
        *qlen = 0;
 
        //wf_coop_iter_impl1<<<(len + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(d_g, d_d, q1, len,q2, qlen, qscratch);
        wf_coop_iter_impl2<block_size><<<(len + block_size - 1) / block_size, block_size>>>(d_g, d_d, q1, len,q2, qlen, qscratch);
        check_cuda(hipMemset(qscratch,0,g.nnodes*sizeof(index_type)));
        hipDeviceSynchronize();

        index_type* tmp = q1;
        q1 = q2;
        q2 = tmp;


        itr += 1;
    }
    end = getTimeStamp();
    double gpu_time = end - start;
    printf("GPU time: %f\n",gpu_time);
}

///////////////////////////////////////////////////////////////////////////////

__global__ void wf_iter_filter(CSRGraph g, edge_data_type* d, index_type* last_q, index_type last_q_len, index_type* touched) {
    index_type index = threadIdx.x + (blockDim.x * blockIdx.x);

    if (index < last_q_len) {
        index_type s_idx = last_q[index];
        for (int j = g.row_start[s_idx]; j < g.row_start[s_idx + 1]; j++) {
            edge_data_type w = d[s_idx];
            edge_data_type ew = g.edge_data[j];
            index_type n = g.edge_dst[j];
            edge_data_type nw = d[n];
            edge_data_type new_w = ew + w;
            // Check if the distance is already set to max then just take the max since,
            if (w >= MAX_VAL){
                new_w = MAX_VAL;
            }

            if (new_w < nw) {
                atomicMin(&d[n],new_w);
                touched[n] = 1;
            }
        }
    }
}

__global__ void setup_id(index_type* out, index_type n) {
    index_type index = threadIdx.x + (blockDim.x * blockIdx.x);
    if (index < n)  out[index] = index;
}

void wf_sweep_filter(CSRGraph& g, CSRGraph& d_g, edge_data_type* d_dists, index_type source) {
    double start,end = 0;
    index_type* q = NULL;
    index_type* scan_indices = NULL;
    index_type* touched = NULL;
    check_cuda(hipMalloc(&q, g.nnodes * sizeof(index_type)));
    check_cuda(hipMalloc(&touched, g.nnodes* sizeof(index_type)));
    check_cuda(hipMalloc(&scan_indices, g.nnodes* sizeof(index_type)));
    setup_id<<<(g.nnodes + 512 - 1),512>>>(scan_indices,g.nnodes);

    // Set first q entry to source
    check_cuda(hipMemcpy(q, &source, sizeof(index_type),hipMemcpyHostToDevice));
    index_type* qlen = NULL;
    check_cuda(hipMallocManaged(&qlen, sizeof(index_type)));
    *qlen = 1;

    void* flg_tmp_store = NULL;
    size_t flg_store_size = 0;
    // index_type num_selected = 0;
    hipcub::DeviceSelect::Flagged(flg_tmp_store,flg_store_size,scan_indices,touched,q,qlen,g.nnodes);
    check_cuda(hipMalloc(&flg_tmp_store,flg_store_size));


    start = getTimeStamp();
    while (*qlen) {
        printf("Iter %d\n",*qlen);
        index_type len = *qlen;
        *qlen = 0;
 
        wf_iter_filter<<<(len + 512 - 1) / 512,512>>>(d_g, d_dists, q, len, touched);
        hipcub::DeviceSelect::Flagged(flg_tmp_store,flg_store_size,scan_indices,touched,q,qlen,g.nnodes);
        check_cuda(hipMemset(touched,0,g.nnodes*sizeof(index_type)));
        hipDeviceSynchronize();
    }
    end = getTimeStamp();
    double gpu_time = end - start;
    printf("GPU time: %f\n",gpu_time);

}

///////////////////////////////////////////////////////////////////////////////////////////

template <int block_size>
__global__ void wf_frontier_kernel(CSRGraph g, edge_data_type* d, index_type* frontier_in, index_type* frontier_out, index_type n, index_type* block_offsets) {
    __shared__ index_type vertices[block_size];
    __shared__ index_type first_edge_offset[block_size];
    __shared__ index_type output_offset[block_size];
    __shared__ uint64_t block_offset[1];

    // Specialize BlockScan for a 1D block of 128 threads on type int
    typedef hipcub::BlockScan<index_type, block_size> BlockScan;
     
    // Allocate shared memory for BlockScan
    __shared__ typename BlockScan::TempStorage temp_storage;

    index_type gidx = threadIdx.x + (blockDim.x * blockIdx.x);
    index_type tidx = threadIdx.x;
    
    index_type degree = 0;
    if (gidx < n) {
        index_type v = frontier_in[gidx];
        if (v != UINT_MAX) {
            vertices[tidx] = v;
            // if (helper[v]) {
            //     printf("BAD! Vertex %d already grabbed!\n",v);
            // }
            // helper[v] = 1;
            index_type row_start =  g.row_start[v];
            first_edge_offset[tidx] = row_start;
            degree = g.row_start[v+1] - row_start;
        } else {
            vertices[tidx] = UINT_MAX;
        }
    }
    // if (gidx == 0) {
    // printf("Thread %d has node %d with degree %d\n",tidx, vertices[tidx], degree);
    // }

    __syncthreads();

    index_type block_aggregate = 0;
    BlockScan(temp_storage).ExclusiveSum(degree, degree, block_aggregate);
    output_offset[tidx] = degree;
    // if (gidx == 0) {
    // printf("Block aggregate %d\n",block_aggregate);
    // }
    __syncthreads();

    if (tidx == 0 && block_aggregate) {
        block_offset[0] = atomicAdd(block_offsets,block_aggregate);
    }
    // if (gidx == 0) {
    // printf("\nBlock totals %d\n",*block_offsets);
    // }


    __syncthreads();

    for (index_type edge_id = tidx; edge_id < block_aggregate; edge_id += block_size) {
        // search for edge
        index_type v_id = 0;
        {
            index_type lo = 0;
            index_type hi = block_size;
            while (lo != hi-1) {
                v_id = lo + (hi - lo)/2;
                if (edge_id >= output_offset[v_id]) {
                    lo = v_id;
                } else {
                    hi = v_id;
                }
            }
            v_id = lo;
        }

        index_type edge_offset = edge_id - output_offset[v_id];
        index_type v_in = vertices[v_id];
        // if (v_in == 0) printf("Got vertex 0! Edge_id %d\n", edge_id);
        index_type edge_dst = g.edge_dst[first_edge_offset[v_id]+ edge_offset];
        edge_data_type ew = g.edge_data[first_edge_offset[v_id]+ edge_offset];
        // printf("exploring edge %d\n",first_edge_offset[v_id]+ edge_offset);
        edge_data_type vw = d[v_in];
        edge_data_type old_dw = d[edge_dst];
        edge_data_type new_dw = vw + ew;

        index_type out_val = UINT_MAX;
        if (new_dw < old_dw) {
            atomicMin(&d[edge_dst], new_dw);
            out_val = edge_dst;
        }
        frontier_out[block_offset[0] + edge_id] = out_val;
        // } else {
        //     frontier_out[block_offset[0] + edge_id] = UINT_MAX;
        // }
        // if (d[edge_dst] == 0) printf("\nEdge ID %d. Using local vert %d (%d)source weight: %d. Dest: %d (%d). new: %d \n", edge_id, v_id, v_in, vw, edge_dst, old_dw, new_dw);

    }
}

__global__ void filter_frontier(index_type* frontier_in, index_type* frontier_out, index_type n, index_type* visited, index_type iteration) {
    index_type gidx = threadIdx.x + (blockDim.x * blockIdx.x);

    if (gidx < n) {
        index_type v = frontier_in[gidx];
        index_type out = UINT_MAX;
        if (v != UINT_MAX) {
            if (atomicExch(&visited[v],iteration) != iteration) {
                out = v;
            } else {
                // printf("Skipping vertex %d\n",v);
            }
        }
        frontier_out[gidx] = out;
    }
}

template <int block_size>
void wf_sweep_frontier(CSRGraph& g, CSRGraph& d_g, edge_data_type* d_dists, index_type source) {
    double start,end = 0;
    index_type* frontier1, *frontier2 = NULL;
    check_cuda(hipMalloc(&frontier1, g.nedges * sizeof(index_type)));
    check_cuda(hipMalloc(&frontier2, g.nedges * sizeof(index_type)));
    index_type* m_N = NULL;
    index_type* visited = NULL;
    check_cuda(hipMalloc(&visited, g.nnodes * sizeof(index_type)));
    check_cuda(hipMallocManaged(&m_N, sizeof(index_type)));
    *m_N = 1;
    check_cuda(hipMemcpy(frontier1,&source, sizeof(index_type), hipMemcpyHostToDevice));


    start = getTimeStamp();

    index_type iter = 0;
    while(*m_N) {
        index_type n = *m_N;
        *m_N = 0;
        printf("Iter %d\n",n);

        wf_frontier_kernel<block_size><<<(n + block_size-1)/block_size,block_size>>>(d_g, d_dists, frontier1, frontier2, n, m_N);
        hipDeviceSynchronize();


        // filter
        n = *m_N;
        filter_frontier<<<(n + block_size-1)/block_size,block_size>>>(frontier2, frontier1,n,visited,iter+1);
        hipDeviceSynchronize();

        iter++;
    }
    end = getTimeStamp();
    double gpu_time = end - start;
    printf("GPU time: %f\n",gpu_time);
}



/////////////////////////////////////////////////////////



void workfront_sweep(CSRGraph& g, edge_data_type* dists, index_type source) {
    CSRGraph d_g;
    g.copy_to_gpu(d_g);
    edge_data_type* d_d = NULL;
    check_cuda(hipMalloc(&d_d, g.nnodes * sizeof(edge_data_type)));
    // Initialize for source node = 0. Otherwise need to change this
    check_cuda(hipMemset(&d_d[0], 0xFF,  (g.nnodes) * sizeof(edge_data_type)));
    check_cuda(hipMemset(&d_d[source], 0,  sizeof(edge_data_type)));

    wf_sweep_frontier<256>(g, d_g, d_d, source);    


    hipMemcpy(dists, d_d, g.nnodes * sizeof(edge_data_type), hipMemcpyDeviceToHost);
}

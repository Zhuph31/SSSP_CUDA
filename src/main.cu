#include "common.cuh"
#include "csr.h"
#include "bellman-ford.cuh"
#include "workfront-sweep.cuh"
#include "nearfar.cuh"
#include "bucketing.cuh"
#include "dijkstra.h"


#include <math.h>





int main(int argc, char** argv) {
    CSRGraph g, gg;
    double start,end = 0;

    if (argc != 2){
        printf("usage program <dataset path>\n");
        return 1; 
    }

    g.read(argv[1]); 
    //g.read("inputs/rmat22.gr");
    // init_trivial_graph(g);

    std::vector<edge_data_type> out_cpu;

    start = getTimeStamp();
    dijkstra(g, out_cpu, 0);
    end = getTimeStamp();
    double cpu_time = end - start;
    printf("CPU time: %f\n",cpu_time);


    edge_data_type* h_d = NULL;
    check_cuda(hipHostMalloc(&h_d, g.nnodes * sizeof(edge_data_type),hipHostMallocWriteCombined));

    start = getTimeStamp();

    // workfront_sweep(g, h_d, 0);
    workfront_sweep_evaluation(g, h_d, 0, out_cpu.data());
    //nearfar(g,h_d);
    
    end = getTimeStamp();
    double gpu_time = end - start;
    printf("Total GPU time: %f\n",gpu_time);


    compare(out_cpu.data(),h_d, out_cpu.size());
    // for (int i = 0; i < 40; i++) {
    //     printf("%d ",h_d[i]);
    // }
    // printf("\n");
    return 0;
}
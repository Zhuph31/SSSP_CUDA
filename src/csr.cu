#include "csr.h"

#include <fstream>
#include <sys/stat.h>
#include <sys/mman.h>

CSRGraph::CSRGraph() {
  init();
}

unsigned CSRGraph::init() {
  row_start = edge_dst = NULL;
  edge_data = NULL;
  node_data = NULL;
  nnodes = nedges = 0;
  device_graph = false;

  return 0;
}

unsigned CSRGraph::read(char file[]) {
  return readFromGR(file);
}


unsigned CSRGraph::readFromGR(char file[]) {
  std::ifstream cfile;
  cfile.open(file);

  // copied from GaloisCpp/trunk/src/FileGraph.h
  int masterFD = open(file, O_RDONLY);
  if (masterFD == -1) {
    printf("FileGraph::structureFromFile: unable to open %s.\n", file);
    return 1;
  }

  struct stat buf;
  int f = fstat(masterFD, &buf);
  if (f == -1) {
    printf("FileGraph::structureFromFile: unable to stat %s.\n", file);
    abort();
  }
  size_t masterLength = buf.st_size;

  int _MAP_BASE = MAP_PRIVATE;
  //#ifdef MAP_POPULATE
  //  _MAP_BASE  |= MAP_POPULATE;
  //#endif

  void* m = mmap(0, masterLength, PROT_READ, _MAP_BASE, masterFD, 0);
  if (m == MAP_FAILED) {
    m = 0;
    printf("FileGraph::structureFromFile: mmap failed.\n");
    abort();
  }

//   ggc::Timer t("graphreader");
//   t.start();

  //parse file
  uint64_t* fptr = (uint64_t*)m;
  __attribute__((unused)) uint64_t version = le64toh(*fptr++);
  assert(version == 1);
  uint64_t sizeEdgeTy = le64toh(*fptr++);
  uint64_t numNodes = le64toh(*fptr++);
  uint64_t numEdges = le64toh(*fptr++);
  uint64_t *outIdx = fptr;
  fptr += numNodes;
  uint32_t *fptr32 = (uint32_t*)fptr;
  uint32_t *outs = fptr32; 
  fptr32 += numEdges;
  if (numEdges % 2) fptr32 += 1;
  unsigned  *edgeData = (unsigned *)fptr32;
	
  // cuda.
  nnodes = numNodes;
  nedges = numEdges;

  printf("nnodes=%d, nedges=%d.\n", nnodes, nedges);
  allocOnHost();

  row_start[0] = 0;

  for (unsigned ii = 0; ii < nnodes; ++ii) {
    row_start[ii+1] = le64toh(outIdx[ii]);
    //   //noutgoing[ii] = le64toh(outIdx[ii]) - le64toh(outIdx[ii - 1]);
    index_type degree = row_start[ii+1] - row_start[ii];

    for (unsigned jj = 0; jj < degree; ++jj) {
      unsigned edgeindex = row_start[ii] + jj;

      unsigned dst = le32toh(outs[edgeindex]);
      if (dst >= nnodes) printf("\tinvalid edge from %d to %d at index %d(%d).\n", ii, dst, jj, edgeindex);

      edge_dst[edgeindex] = dst;

      if(sizeEdgeTy)
	edge_data[edgeindex] = edgeData[edgeindex];
    }

    progressPrint(nnodes, ii);
  }

  cfile.close();	// probably galois doesn't close its file due to mmap.
//   t.stop();

  // TODO: fix MB/s
//   printf("read %lld bytes in %d ms (%0.2f MB/s)\n\r\n", masterLength, t.duration_ms(), (masterLength / 1000.0) / (t.duration_ms()));

  return 0;
}

void CSRGraph::progressPrint(unsigned maxii, unsigned ii) {
  const unsigned nsteps = 10;
  unsigned ineachstep = (maxii / nsteps);
  if(ineachstep == 0) ineachstep = 1;
  /*if (ii == maxii) {
    printf("\t100%%\n");
    } else*/ if (ii % ineachstep == 0) {
    int progress = ((size_t) ii * 100) / maxii + 1;

    printf("\t%3d%%\r", progress);
    fflush(stdout);
  }
}

unsigned CSRGraph::allocOnHost() {
  assert(nnodes > 0);
  assert(!device_graph);

  if(row_start != NULL) // already allocated
    return true;

  row_start = (index_type *) calloc(nnodes+1, sizeof(index_type));
  edge_dst  = (index_type *) calloc(nedges, sizeof(index_type));
  edge_data = (edge_data_type *) calloc(nedges, sizeof(edge_data_type));
  node_data = (node_data_type *) calloc(nnodes, sizeof(node_data_type));

  size_t mem_usage = ((nnodes + 1) + nedges) * sizeof(index_type) 
    + (nedges) * sizeof(edge_data_type) + (nnodes) * sizeof(node_data_type);
    
  printf("Host memory for graph: %3u MB\n", mem_usage / 1048756);

  return (edge_data && row_start && edge_dst && node_data);
}


unsigned CSRGraph::allocOnDevice() {
  if(edge_dst != NULL)  // already allocated
    return true;  

  assert(edge_dst == NULL); // make sure not already allocated

  check_cuda(hipMalloc((void **) &edge_dst, nedges * sizeof(index_type)));
  check_cuda(hipMalloc((void **) &row_start, (nnodes+1) * sizeof(index_type)));

  check_cuda(hipMalloc((void **) &edge_data, nedges * sizeof(edge_data_type)));
  check_cuda(hipMalloc((void **) &node_data, nnodes * sizeof(node_data_type)));

  device_graph = true;

  return (edge_dst && edge_data && row_start && node_data);
}

void CSRGraph::copy_to_gpu(struct CSRGraph &copygraph) {
  copygraph.nnodes = nnodes;
  copygraph.nedges = nedges;
  
  assert(copygraph.allocOnDevice());

  check_cuda(hipMemcpy(copygraph.edge_dst, edge_dst, nedges * sizeof(index_type), hipMemcpyHostToDevice));
  check_cuda(hipMemcpy(copygraph.edge_data, edge_data, nedges * sizeof(edge_data_type), hipMemcpyHostToDevice));
  check_cuda(hipMemcpy(copygraph.node_data, node_data, nnodes * sizeof(edge_data_type), hipMemcpyHostToDevice));

  check_cuda(hipMemcpy(copygraph.row_start, row_start, (nnodes+1) * sizeof(index_type), hipMemcpyHostToDevice));
}


void init_trivial_graph(CSRGraph& g) {
    g.nnodes = 6;
    g.nedges = 16;
    g.row_start = (index_type *)malloc((g.nnodes+1)*sizeof(index_type));
    g.edge_dst = (index_type *)malloc(g.nedges*sizeof(edge_data_type));
    g.edge_data = (edge_data_type *)malloc(g.nedges*sizeof(edge_data_type));
    g.node_data = (node_data_type *)malloc(g.nnodes*sizeof(edge_data_type));

    g.row_start[0] = 0;
    g.row_start[1] = 2;
    g.row_start[2] = 5;
    g.row_start[3] = 9;
    g.row_start[4] = 11;
    g.row_start[5] = 13;
    g.row_start[6] = 16;

    g.edge_dst[0] = 1;
    g.edge_dst[1] = 2;

    g.edge_dst[2] = 0;
    g.edge_dst[3] = 2;
    g.edge_dst[4] = 3;

    g.edge_dst[5] = 0;
    g.edge_dst[6] = 1;
    g.edge_dst[7] = 4;
    g.edge_dst[8] = 5;

    g.edge_dst[9] = 1;
    g.edge_dst[10] = 5;

    g.edge_dst[11] = 2;
    g.edge_dst[12] = 5;

    g.edge_dst[13] = 2;
    g.edge_dst[14] = 3;
    g.edge_dst[15] = 4;

    g.edge_data[0] = 2;
    g.edge_data[1] = 4;

    g.edge_data[2] = 2;
    g.edge_data[3] = 1;
    g.edge_data[4] = 6;

    g.edge_data[5] = 4;
    g.edge_data[6] = 1;
    g.edge_data[7] = 2;
    g.edge_data[8] = 3;

    g.edge_data[9] = 6;
    g.edge_data[10] = 5;

    g.edge_data[11] = 2;
    g.edge_data[12] = 4;

    g.edge_data[13] = 3;
    g.edge_data[14] = 5;
    g.edge_data[15] = 4;
}

